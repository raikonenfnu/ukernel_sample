
#include <hip/hip_runtime.h>
// Copyright 2022 The IREE Authors
//
// Licensed under the Apache License v2.0 with LLVM Exceptions.
// See https://llvm.org/LICENSE.txt for license information.
// SPDX-License-Identifier: Apache-2.0 WITH LLVM-exception

// This minimal example just has some publicly exported (__global__) kernels.
// It's possible with more build goo to include .cuh files and pull in any
// CUDA functions that do not involve host behavior (kernel launches/etc).
//
// NOTE: kernels must be exported with C naming (no C++ mangling) in order to
// match the names used in the IR declarations.
//
// NOTE: arguments are packed as a dense list of
// ([ordered bindings...], [push constants...]). If a binding is declared as
// read-only the kernel must not write to it as it may be shared by other
// invocations.
//
// NOTE: today all constants must be i32. If larger types are required there are
// packing rules that must line up with compiler expectations - passed i64
// values must be padded to natural 8-byte alignment, for example.
//
// NOTE: IREE ensures that all I/O buffers are legal to have the __restrict__
// keyword defined (no aliasing is induced that is potentially unsafe). It's
// still possible for users to do bad things but such is the case with native
// CUDA programming.
//
// NOTE: I/O buffer base pointers are likely to be nicely aligned (64B minimum
// but usually larger) but the pointers passed in may be offset by any value
// as they represent subranges of the underlying buffers. For example if the
// user slices out elements 3 and 4 out of a 4xf32 tensor then the base buffer
// pointer will be at +8B. In general if the input wasn't trying to be tricky
// (bitcasting/etc) then natural alignment is guaranteed (an f32 tensor will
// always have buffer pointers aligned to 4B).

// `ret = lhs * rhs`
//
// Conforms to ABI:
// #hal.pipeline.layout<push_constants = 1, sets = [
//   <0, bindings = [
//       <0, storage_buffer, ReadOnly>,
//       <1, storage_buffer, ReadOnly>,
//       <2, storage_buffer>
//   ]>
// ]>
// workgroup_size = [64 : index, 1 : index, 1 : index]
extern "C" __global__ void simple_mul(const float* __restrict__ binding0,
                                      const float* __restrict__ binding1,
                                      float* __restrict__ binding2, int dim) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < dim) {
    binding2[tid] = binding0[tid] * binding1[tid];
  }
}

// `rhs *= lhs`
//
// Conforms to ABI:
// #hal.pipeline.layout<push_constants = 1, sets = [
//   <0, bindings = [
//       <0, storage_buffer, ReadOnly>,
//       <1, storage_buffer>
//   ]>
// ]>
// workgroup_size = [64 : index, 1 : index, 1 : index]
extern "C" __global__ void simple_mul_inplace(
    const float* __restrict__ binding0, float* __restrict__ binding1, int dim) {
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid < dim) {
    binding1[tid] *= binding0[tid];
  }
}
